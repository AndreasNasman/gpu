
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#define BIN_WIDTH 0.25
#define BLOCK_DIM 256
#define LINE_LENGTH 30

typedef struct Galaxy
{
    float declination;
    float right_ascension;
} Galaxy;

typedef struct GalaxySet
{
    Galaxy *galaxies;
} GalaxySet;

__global__ void build_histograms(GalaxySet real, GalaxySet random, int *DD_histogram, int *DR_histogram, int *RR_histogram, int n);
__global__ void galaxy_distribution(int *DD_histogram, int *DR_histogram, int *RR_histogram, float *distribution, int n);
void read_file(FILE *file_pointer, const char *DELIMITER, Galaxy *galaxy_set, int n);
void write_file_int(FILE *file_pointer, int *content, int n);
void write_file_float(FILE *file_pointer, float *content, int n);

int main()
{
    /* READING REAL GALAXIES FILE */
    FILE *file_pointer = fopen("./input-data/real-galaxies.txt", "r");
    const char *DELIMITER = "\t";

    // Reads number of lines to process (defined on the first line of the file).
    char line[LINE_LENGTH];
    fgets(line, LINE_LENGTH, file_pointer);
    const int NUMBER_OF_LINES = atoi(line);

    GalaxySet real;
    hipMallocManaged(&real.galaxies, NUMBER_OF_LINES * sizeof(Galaxy));

    read_file(file_pointer, DELIMITER, real.galaxies, NUMBER_OF_LINES);

    /* READING RANDOM GALAXIES FILE */
    file_pointer = fopen("./input-data/random-galaxies.txt", "r");
    DELIMITER = " ";

    // Checks that number of lines is equal in both files.
    fgets(line, LINE_LENGTH, file_pointer);
    if (NUMBER_OF_LINES != atoi(line))
    {
        printf("Both files should have equal number of lines!");
        return 1;
    }

    GalaxySet random;
    hipMallocManaged(&random.galaxies, NUMBER_OF_LINES * sizeof(Galaxy));

    read_file(file_pointer, DELIMITER, random.galaxies, NUMBER_OF_LINES);

    /* BUILDING HISTOGRAMS */
    const int COVERAGE = 180; // degrees
    const int NUMBER_OF_BINS = COVERAGE * (1 / BIN_WIDTH);

    // Defines number of blocks to use.
    const int GRID_DIM = ceil(NUMBER_OF_LINES / (float)BLOCK_DIM);

    int *DD_histogram_temp, *DR_histogram_temp, *RR_histogram_temp;
    hipMallocManaged(&DD_histogram_temp, NUMBER_OF_BINS * GRID_DIM * sizeof(int));
    hipMallocManaged(&DR_histogram_temp, NUMBER_OF_BINS * GRID_DIM * sizeof(int));
    hipMallocManaged(&RR_histogram_temp, NUMBER_OF_BINS * GRID_DIM * sizeof(int));

    build_histograms<<<GRID_DIM, BLOCK_DIM>>>(real, random, DD_histogram_temp, DR_histogram_temp, RR_histogram_temp, NUMBER_OF_LINES);
    hipDeviceSynchronize();

    int *DD_histogram, *DR_histogram, *RR_histogram;
    hipMallocManaged(&DD_histogram, NUMBER_OF_BINS * sizeof(int));
    hipMallocManaged(&DR_histogram, NUMBER_OF_BINS * sizeof(int));
    hipMallocManaged(&RR_histogram, NUMBER_OF_BINS * sizeof(int));

    for (int i = 0; i < NUMBER_OF_BINS * GRID_DIM; i += 1)
    {
        DD_histogram[i % NUMBER_OF_BINS] += DD_histogram_temp[i];
        DR_histogram[i % NUMBER_OF_BINS] += DR_histogram_temp[i];
        RR_histogram[i % NUMBER_OF_BINS] += RR_histogram_temp[i];
    }

    /* DETERMINING DISTRIBUTION */
    float *distribution;
    hipMallocManaged(&distribution, NUMBER_OF_BINS * sizeof(float));

    galaxy_distribution<<<GRID_DIM, BLOCK_DIM>>>(DD_histogram, DR_histogram, RR_histogram, distribution, NUMBER_OF_BINS);
    hipDeviceSynchronize();

    /* WRITING RESULTS TO FILE */
    system("mkdir -p results");

    file_pointer = fopen("results/DD_histogram.txt", "w");
    write_file_int(file_pointer, DD_histogram, NUMBER_OF_BINS);

    file_pointer = fopen("results/RR_histogram.txt", "w");
    write_file_int(file_pointer, RR_histogram, NUMBER_OF_BINS);

    file_pointer = fopen("results/Distribution.txt", "w");
    write_file_float(file_pointer, distribution, NUMBER_OF_BINS);

    /* CLEAN UP */
    fclose(file_pointer);

    hipFree(real.galaxies);
    hipFree(random.galaxies);
    hipFree(DD_histogram);
    hipFree(DR_histogram);
    hipFree(RR_histogram);

    printf("Done!\n\n");

    return 0;
}

__device__ float angle_between_two_galaxies(Galaxy first_galaxy, Galaxy second_galaxy)
{
    float x = sinf(first_galaxy.declination) * sinf(second_galaxy.declination) +
              cosf(first_galaxy.declination) * cosf(second_galaxy.declination) *
                  cosf(first_galaxy.right_ascension - second_galaxy.right_ascension);

    // Checks that x is within the boundaries of [-1.0f, 1.0f].
    return acosf(fmin(1.0f, fmax(-1.0f, x)));
}

__device__ float radians_to_degrees(float radian_value)
{
    return radian_value * (180 / M_PI);
}

__device__ void update_bin(int *bin, float angle, int incrementor)
{
    int index = floor(radians_to_degrees(angle) / BIN_WIDTH);
    atomicAdd(&bin[index], incrementor);
}

__global__ void build_histograms(GalaxySet real, GalaxySet random, int *DD_histogram, int *DR_histogram, int *RR_histogram, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    __shared__ int shared_DD_histogram[720];
    __shared__ int shared_DR_histogram[720];
    __shared__ int shared_RR_histogram[720];
    for (int i = 0; i < 720; i += 1)
    {
        shared_DD_histogram[i] = 0;
        shared_DR_histogram[i] = 0;
        shared_RR_histogram[i] = 0;
    }
    __syncthreads();

    float angle;
    for (int i = 0; i < n; i += 1)
        for (int j = index; j < n; j += stride)
        {
            // Every pair of real-random galaxy is compared.
            angle = angle_between_two_galaxies(real.galaxies[i], random.galaxies[j]);
            update_bin(shared_DR_histogram, angle, 1);

            // Real-real and random-random galaxy pairs are only compared from the same starting index forward.
            // If both indexes are the same, the relevant bin is incremented by one.
            if (j == i)
            {
                angle = 0;
                update_bin(shared_DD_histogram, angle, 1);
                update_bin(shared_RR_histogram, angle, 1);
            }
            // When one of the indexes is greater, the relevant bin is incremented by two.
            // This is the same as doing the comparison twice, thus saving execution time.
            else if (j > i)
            {
                angle = angle_between_two_galaxies(real.galaxies[i], real.galaxies[j]);
                update_bin(shared_DD_histogram, angle, 2);

                angle = angle_between_two_galaxies(random.galaxies[i], random.galaxies[j]);
                update_bin(shared_RR_histogram, angle, 2);
            }
        }
    __syncthreads();

    for (int i = 0; i < 720; i += 1)
    {
        DD_histogram[720 * blockIdx.x + i] = shared_DD_histogram[i];
        DR_histogram[720 * blockIdx.x + i] = shared_DR_histogram[i];
        RR_histogram[720 * blockIdx.x + i] = shared_RR_histogram[i];
    }
}

__global__ void galaxy_distribution(int *DD_histogram, int *DR_histogram, int *RR_histogram, float *distribution, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride)
    {
        if (RR_histogram[i] == 0)
            continue;

        distribution[i] = (DD_histogram[i] - 2.0f * DR_histogram[i] + RR_histogram[i]) / RR_histogram[i];
    }
}

float arcminutes_to_radians(float arcminute_value)
{
    return (M_PI * arcminute_value) / (60 * 180);
}

void read_file(FILE *file_pointer, const char *DELIMITER, Galaxy *galaxies, int n)
{
    char line[LINE_LENGTH];
    const int DECLINATION_INDEX = 1;
    const int RIGHT_ASCENSION_INDEX = 0;

    for (int i = 0; i < n; i += 1)
    {
        fgets(line, LINE_LENGTH, file_pointer);

        char *token = strtok(line, DELIMITER);

        int index = 0;
        while (token != NULL)
        {
            float arcminute_value = atof(token);

            if (index == DECLINATION_INDEX)
                galaxies[i].declination = arcminutes_to_radians(arcminute_value);
            else if (index == RIGHT_ASCENSION_INDEX)
                galaxies[i].right_ascension = arcminutes_to_radians(arcminute_value);

            index += 1;
            token = strtok(NULL, DELIMITER);
        }
    }
}

void write_file_int(FILE *file_pointer, int *content, int n)
{
    for (int i = 0; i < n; i += 1)
        fprintf(file_pointer, "%d\n", content[i]);
}

void write_file_float(FILE *file_pointer, float *content, int n)
{
    for (int i = 0; i < n; i += 1)
        fprintf(file_pointer, "%f\n", content[i]);
}
